#include "hip/hip_runtime.h"
/*!
 *  \file       bvh_tree.cu
 *  \brief      
 *  
 */


#include "bvh_tree.cuh"


__global__ void findPotentialCollisions(CollisionList   list,
                                        BVH             bvh,
                                        AABB*           objectAABBs,
                                        int             numObjects)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < numObjects)
    {
        traverseRecursive(list, bvh, objectAABBs[idx], idx, bvh.getRoot());
    }
}

__device__ void traverseIterative(CollisionList&    list,
                                  BVH&              bvh,
                                  AABB&             queryAABB,
                                  int               queryObjectIdx)
{
    /*  Allocate traversal stack from thread-local memory, and push nullptr
     *  to indicate that there are no postponed nodes.
     */
    NodePtr stack[6];
    NodePtr* stackPtr = stack;
    *stackPtr++ = nullptr;  // push stack

    // Traverse nodes starting from the root.
    NodePtr node = bvh.getRoot();
    do
    {
        // Check each child node for overlap
        NodePtr left_child = bvh.getLeftChild(node);
        NodePtr right_child = bvh.getRightChild(node);
        bool left_overlap = checkOverlap(queryAABB, bvh.getAABB(left_child));
        bool right_overlap = checkOverlap(queryAABB, bvh.getAABB(right_child));

        // Query whether to overlap with a leaf node, in order to report collision.
        if (left_overlap && bvh.isLeaf(left_child))
        {
            list.add(queryObjectIdx, bvh.getObjectIdx(left_child));
        }
        if (right_overlap && bvh.isLeaf(right_child))
        {
            list.add(queryObjectIdx, bvh.getObjectIdx(right_child));
        }

        // Query whether to overlap with an internal node, then traverse recursively
        bool left_traverse = (left_overlap && !bvh.isLeaf(left_child));
        bool right_traverse = (right_overlap && !bvh.isLeaf(right_child));
        if (!left_traverse and !right_overlap)
        {
            node = *--stackPtr;     // pop stack
        }
        else
        {
            node = left_traverse ? left_child : right_child;
            if (left_traverse && right_traverse)
            {
                *stackPtr++ = right_child;  // push stack
            }
        }
    }
    while (node != nullptr);
}

__global__ void findPotentialCollisions(CollisionList   list,
                                        BVH             bvh)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < bvh.getNumLeaves())
    {
        NodePtr leaf = bvh.getLeaf(idx);
        AABB leaf_aabb = bvh.getAABB(leaf);
        traverseIterative(list, bvh, leaf_aabb, bvh.getObjectIdx(leaf));
    }
}